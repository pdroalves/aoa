#include <stdlib.h>
#include <fstream>
#include <iterator>
#include <iomanip>
#include <hip/hip_runtime_api.h>
#include <NTL/ZZ.h>
#include <NTL/ZZ_pX.h>
#include <NTL/ZZ_pE.h>
#include <time.h>
#include <unistd.h>
#include <iomanip>
#include <AOADGT/settings.h>
#include <AOADGT/arithmetic/polynomial.h>
#include <AOADGT/cuda/sampler.h>
#include <cxxopts.hpp>
#include <hip/hip_runtime_api.h>
#include <AOADGT/ckks.h>
#include <AOADGT/ckkscontext.h>
#include <AOADGT/tool/version.h>
#include <AOADGT/tool/version.h>

#define BILLION  1000000000L
#define MILLION  1000000L
#define NITERATIONS 100

__host__ double compute_time_ms(struct timespec start,struct timespec stop){
  return (( stop.tv_sec - start.tv_sec )*BILLION + ( stop.tv_nsec - start.tv_nsec ))/MILLION;
}

double print_memory_usage(){
  // show memory usage of GPU
  size_t free_byte ; size_t total_byte ;
  hipMemGetInfo( &free_byte, &total_byte ) ;
  double free_db = (double)free_byte ;
  double total_db = (double)total_byte ;
  double used_db = total_db - free_db ;

  std::cout << "GPU memory usage: used = " << used_db/1024.0/1024.0 << 
  ", free = " << free_db/1024.0/1024.0 << " MB, total = " <<
  total_db/1024.0/1024.0 << " MB\n" << std::endl;

  return free_db/1024.0/1024.0;
}

__host__ double runInit(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Init
  Context *ctx = new Context();
  poly_t a[NITERATIONS];

  // Warm-up
  poly_init(ctx, &a[0]); 
  poly_free(ctx, &a[0]); 

  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    poly_init(ctx, &a[i]);
  hipEventRecord(stop, ctx->get_stream());

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);
  
  for(int i = 0; i < NITERATIONS; i++)
    poly_free(ctx, &a[i]);
  delete ctx;
  return latency / NITERATIONS;
}

__host__ double runCudaStreamInit(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  hipStream_t *streams;
  streams = (hipStream_t*)malloc(NITERATIONS*sizeof(hipStream_t));

  // Warm-up
  hipStreamCreate(&streams[0]);
  hipStreamDestroy(streams[0]);
  
  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start);
  for(int i = 0; i < NITERATIONS; i++)
    hipStreamCreate(&streams[i]);
  hipEventRecord(stop);

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);
  
  for(int i = 0; i < NITERATIONS; i++)
    hipStreamDestroy(streams[i]);
  free(streams);
  delete ctx;
  return latency / NITERATIONS;
}


__host__ double runPolyAdd(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  Sampler::init(ctx);
  std::vector<poly_t> a(NITERATIONS);
  std::vector<poly_t> b(NITERATIONS);
  std::vector<poly_t> c(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++){
    poly_init(ctx, &a[i]);
    poly_init(ctx, &b[i]);
    poly_init(ctx, &c[i]);
    Sampler::sample(ctx, &a[i], DISCRETE_GAUSSIAN);
    Sampler::sample(ctx, &b[i], DISCRETE_GAUSSIAN);
  }

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    poly_add(ctx, &c[i], &a[i], &b[i]);
  
  // Benchmark
  hipDeviceSynchronize();
  hipProfilerStart();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    poly_add(ctx, &c[i], &a[i], &b[i]);
  hipEventRecord(stop, ctx->get_stream());
  hipEventSynchronize(stop);
  hipProfilerStop();
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++){
    poly_free(ctx, &a[i]);
    poly_free(ctx, &b[i]);
    poly_free(ctx, &c[i]);
  }
  delete ctx;
  return latency / NITERATIONS;
}

__host__ double runPolyMul(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  Sampler::init(ctx);
  std::vector<poly_t> a(NITERATIONS);
  std::vector<poly_t> b(NITERATIONS);
  std::vector<poly_t> c(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++){
    poly_init(ctx, &a[i]);
    poly_init(ctx, &b[i]);
    poly_init(ctx, &c[i]);
    Sampler::sample(ctx, &a[i], DISCRETE_GAUSSIAN);
    Sampler::sample(ctx, &b[i], DISCRETE_GAUSSIAN);
  }

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    poly_mul(ctx, &c[0], &a[0], &b[0]);

  // Benchmark
  hipDeviceSynchronize();
  hipProfilerStart();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    poly_mul(ctx, &c[i], &a[i], &b[i]);
  hipEventRecord(stop, ctx->get_stream());
  hipProfilerStop();
  hipEventSynchronize(stop);

  //
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++){
    poly_free(ctx, &a[i]);
    poly_free(ctx, &b[i]);
    poly_free(ctx, &c[i]);
  }
  delete ctx;
  return latency / NITERATIONS;
}

__host__ double runPolyMulAdd(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  Sampler::init(ctx);
  std::vector<poly_t> a(NITERATIONS);
  std::vector<poly_t> b(NITERATIONS);
  std::vector<poly_t> c(NITERATIONS);
  std::vector<poly_t> d(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++){
    poly_init(ctx, &a[i]);
    poly_init(ctx, &b[i]);
    poly_init(ctx, &c[i]);
    poly_init(ctx, &d[i]);
    Sampler::sample(ctx, &a[i], DISCRETE_GAUSSIAN);
    Sampler::sample(ctx, &b[i], DISCRETE_GAUSSIAN);
    Sampler::sample(ctx, &c[i], DISCRETE_GAUSSIAN);
  }

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    poly_mul_add(ctx, &d[0], &a[0], &b[0], &c[0]);

  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    poly_mul_add(ctx, &d[i], &a[i], &b[i], &c[i]);
  hipEventRecord(stop, ctx->get_stream());

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++){
    poly_free(ctx, &a[i]);
    poly_free(ctx, &b[i]);
    poly_free(ctx, &c[i]);
    poly_free(ctx, &d[i]);
  }
  delete ctx;
  return latency / NITERATIONS;
}

__host__ double runPolyAddAdd(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  Sampler::init(ctx);
  std::vector<poly_t> a(NITERATIONS);
  std::vector<poly_t> b(NITERATIONS);
  std::vector<poly_t> c(NITERATIONS);
  std::vector<poly_t> d(NITERATIONS);
  std::vector<poly_t> e(NITERATIONS);
  std::vector<poly_t> f(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++){
    poly_init(ctx, &a[i]);
    poly_init(ctx, &b[i]);
    poly_init(ctx, &c[i]);
    poly_init(ctx, &d[i]);
    poly_init(ctx, &e[i]);
    poly_init(ctx, &f[i]);
    Sampler::sample(ctx, &a[i], DISCRETE_GAUSSIAN);
    Sampler::sample(ctx, &b[i], DISCRETE_GAUSSIAN);
    Sampler::sample(ctx, &c[i], DISCRETE_GAUSSIAN);
    Sampler::sample(ctx, &d[i], DISCRETE_GAUSSIAN);
    Sampler::sample(ctx, &e[i], DISCRETE_GAUSSIAN);
    Sampler::sample(ctx, &f[i], DISCRETE_GAUSSIAN);
  }

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    poly_double_add(ctx, &a[0], &b[0], &c[0], &d[0], &e[0], &f[0]);

  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    poly_double_add(ctx, &a[i], &b[i], &c[i], &d[i], &e[i], &f[i]);
  hipEventRecord(stop, ctx->get_stream());

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++){
    poly_free(ctx, &a[i]);
    poly_free(ctx, &b[i]);
    poly_free(ctx, &c[i]);
    poly_free(ctx, &d[i]);
    poly_free(ctx, &e[i]);
    poly_free(ctx, &f[i]);
  }
  delete ctx;
  return latency / NITERATIONS;
}

double runAdd(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  std::default_random_engine generator;
  std::uniform_real_distribution<double> distribution;

  float latency = 0;

  // Setup
  CKKSContext *cipher = new CKKSContext();
  Sampler::init(cipher);
  SecretKey *sk = ckks_new_sk(cipher);
  Keys *keys = ckks_keygen(cipher, sk);

  complex<double> val1 = {distribution(generator), distribution(generator)};
  complex<double> val2 = {distribution(generator), distribution(generator)};

  std::vector<cipher_t*> ct1;
  std::vector<cipher_t*> ct2;
  std::vector<cipher_t*> ct3;
  
  for(int i = 0; i < NITERATIONS; i++){
    ct1.push_back(ckks_encrypt(cipher, &val1));
    ct2.push_back(ckks_encrypt(cipher, &val2));
    cipher_t *c = new cipher_t;
    cipher_init(cipher, c);
    ct3.push_back(c);
  }

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    ckks_add(cipher, ct3[i], ct1[i], ct2[i]);

  // Benchmark
  hipDeviceSynchronize();
  cudaCheckError();
  hipProfilerStart();
  hipEventRecord(start, cipher->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
      ckks_add(cipher, ct3[i], ct1[i], ct2[i]);  

  hipEventRecord(stop, cipher->get_stream());
  cudaCheckError();
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);
  hipProfilerStop();

  // Release memory  
  for(int i = 0; i < NITERATIONS; i++){
    cipher_free(cipher, ct1[i]);
    cipher_free(cipher, ct2[i]);
    cipher_free(cipher, ct3[i]);
  }
    
  keys_free(cipher, keys);
  poly_free(cipher, &sk->s);
  delete sk;
  delete keys;
  delete cipher;
  return latency / NITERATIONS;
}

__host__ double runMul(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  std::default_random_engine generator;
  std::uniform_real_distribution<double> distribution;

  float latency = 0;

  // Setup
  CKKSContext *cipher = new CKKSContext();
  Sampler::init(cipher);
  SecretKey *sk = ckks_new_sk(cipher);
  Keys *keys = ckks_keygen(cipher, sk);

  complex<double> val1 = {distribution(generator), distribution(generator)};
  complex<double> val2 = {distribution(generator), distribution(generator)};

  std::vector<cipher_t*> ct1;
  std::vector<cipher_t*> ct2;
  std::vector<cipher_t*> ct3;
  
  for(int i = 0; i < NITERATIONS; i++){
    ct1.push_back(ckks_encrypt(cipher, &val1));
    ct2.push_back(ckks_encrypt(cipher, &val2));
    cipher_t *c = new cipher_t;
    cipher_init(cipher, c);
    ct3.push_back(c);
  }

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    ckks_mul_without_rescale(cipher, ct3[i], ct1[i], ct2[i]);

  // Benchmark
  hipDeviceSynchronize();
  cudaCheckError();
  hipProfilerStart();
  hipEventRecord(start, cipher->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
      ckks_mul_without_rescale(cipher, ct3[i], ct1[i], ct2[i]);  

  hipEventRecord(stop, cipher->get_stream());
  cudaCheckError();
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);
  
  hipProfilerStop();
  cudaCheckError();

  // Release memory  
  for(int i = 0; i < NITERATIONS; i++){
    cipher_free(cipher, ct1[i]);
    cipher_free(cipher, ct2[i]);
    cipher_free(cipher, ct3[i]);
  }
    
  keys_free(cipher, keys);
  poly_free(cipher, &sk->s);
  delete sk;
  delete keys;
  delete cipher;
  return latency / NITERATIONS;
}

__host__ double runRescale(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  std::default_random_engine generator;
  std::uniform_real_distribution<double> distribution;

  float latency = 0;

  // Setup
  CKKSContext *cipher = new CKKSContext();
  Sampler::init(cipher);
  SecretKey *sk = ckks_new_sk(cipher);
  Keys *keys = ckks_keygen(cipher, sk);

  complex<double> val = {distribution(generator), distribution(generator)};

  std::vector<cipher_t*> ct;
  
  for(int i = 0; i < NITERATIONS; i++){
    cipher_t *c = ckks_encrypt(cipher, &val);

    DGTEngine::execute_dgt(cipher, &c->c[0], INVERSE);
    DGTEngine::execute_dgt(cipher, &c->c[1], INVERSE);

    ct.push_back(c);
  }


  // Benchmark
  hipDeviceSynchronize();
  cudaCheckError();
  hipProfilerStart();
  hipEventRecord(start, cipher->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    ckks_rescale(cipher, ct[i]);

  hipEventRecord(stop, cipher->get_stream());
  cudaCheckError();
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);
  hipProfilerStop();

  // Release memory  
  for(int i = 0; i < NITERATIONS; i++)
    cipher_free(cipher, ct[i]);
    
  keys_free(cipher, keys);
  poly_free(cipher, &sk->s);
  delete sk;
  delete keys;
  delete cipher;
  return latency / NITERATIONS;
}

__host__ double runDGT(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  Sampler::init(ctx);
  std::vector<poly_t> d_data(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++){
    poly_init(ctx, &d_data[i]);
    Sampler::sample(ctx, &d_data[i], DISCRETE_GAUSSIAN);
    DGTEngine::execute_dgt(ctx, &d_data[i], INVERSE);
  }


  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start, ctx->get_stream());
  hipProfilerStart();
  for(int i = 0; i < NITERATIONS; i++)
    DGTEngine::execute_dgt(ctx, &d_data[i], FORWARD);
  hipProfilerStop();
  hipEventRecord(stop, ctx->get_stream());

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++)
    poly_free(ctx, &d_data[i]);
  delete ctx;
  return latency / NITERATIONS;
}

__host__ double runIDGT(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  std::vector<poly_t> d_data(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++){
    poly_init(ctx, &d_data[i]);
    Sampler::sample(ctx, &d_data[i], DISCRETE_GAUSSIAN);
  }

  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    DGTEngine::execute_dgt(ctx, &d_data[i], INVERSE);
  hipEventRecord(stop, ctx->get_stream());

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++)
    poly_free(ctx, &d_data[i]);
  delete ctx;
  return latency / NITERATIONS;
}

__host__ double runRotate(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  std::default_random_engine generator;
  std::uniform_real_distribution<double> distribution;

  float latency = 0;

  // Setup
  CKKSContext *cipher = new CKKSContext();
  Sampler::init(cipher);
  SecretKey *sk = ckks_new_sk(cipher);
  Keys *keys = ckks_keygen(cipher, sk);

  complex<double> val = {distribution(generator), distribution(generator)};

  std::vector<cipher_t*> ct1;
  std::vector<cipher_t*> ct2;
  
  for(int i = 0; i < NITERATIONS; i++){
    ct1.push_back(ckks_encrypt(cipher, &val, 1, CUDAEngine::N - 1));
    cipher_t *c = new cipher_t;
    cipher_init(cipher, c);
    ct2.push_back(c);
  }


  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    ckks_rotate_left(cipher, ct2[i], ct1[i], 1);

  // Benchmark
  hipDeviceSynchronize();
  cudaCheckError();
  hipProfilerStart();
  hipEventRecord(start, cipher->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    ckks_rotate_left(cipher, ct2[i], ct1[i], 1);

  hipEventRecord(stop, cipher->get_stream());
  cudaCheckError();
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);
  hipProfilerStop();

  // Release memory  
  for(int i = 0; i < NITERATIONS; i++){
    cipher_free(cipher, ct1[i]);
    cipher_free(cipher, ct2[i]);
  }
    
  keys_free(cipher, keys);
  poly_free(cipher, &sk->s);
  delete sk;
  delete keys;
  delete cipher;
  return latency / NITERATIONS;
}

__host__ double runSumslots(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  std::default_random_engine generator;
  std::uniform_real_distribution<double> distribution;

  float latency = 0;

  // Setup
  CKKSContext *cipher = new CKKSContext();
  Sampler::init(cipher);
  SecretKey *sk = ckks_new_sk(cipher);
  Keys *keys = ckks_keygen(cipher, sk);

  complex<double> val = {distribution(generator), distribution(generator)};

  std::vector<cipher_t*> ct;
  
  for(int i = 0; i < NITERATIONS; i++){
    ct.push_back(ckks_encrypt(cipher, &val, 1, CUDAEngine::N - 1));
  }


  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    ckks_sumslots(cipher, ct[i], ct[i]);

  // Benchmark
  hipDeviceSynchronize();
  cudaCheckError();
  hipProfilerStart();
  hipEventRecord(start, cipher->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    ckks_sumslots(cipher, ct[i], ct[i]);
  hipEventRecord(stop, cipher->get_stream());
  cudaCheckError();
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);
  hipProfilerStop();

  // Release memory  
  for(int i = 0; i < NITERATIONS; i++)
    cipher_free(cipher, ct[i]);
    
  keys_free(cipher, keys);
  poly_free(cipher, &sk->s);
  delete sk;
  delete keys;
  delete cipher;
  return latency / NITERATIONS;
}

double runAddPlain(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  std::default_random_engine generator;
  std::uniform_real_distribution<double> distribution;

  float latency = 0;

  // Setup
  CKKSContext *cipher = new CKKSContext();
  Sampler::init(cipher);
  SecretKey *sk = ckks_new_sk(cipher);
  Keys *keys = ckks_keygen(cipher, sk);

  complex<double> val1 = {distribution(generator), distribution(generator)};
  double val2 = distribution(generator);

  std::vector<cipher_t*> ct1;
  std::vector<cipher_t*> ct3;
  
  for(int i = 0; i < NITERATIONS; i++){
    ct1.push_back(ckks_encrypt(cipher, &val1));
    cipher_t *c = new cipher_t;
    cipher_init(cipher, c);
    ct3.push_back(c);
  }

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    ckks_add(cipher, ct3[i], ct1[i], val2);

  // Benchmark
  hipDeviceSynchronize();
  cudaCheckError();
  hipProfilerStart();
  hipEventRecord(start, cipher->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
      ckks_add(cipher, ct3[i], ct1[i], val2);  

  hipEventRecord(stop, cipher->get_stream());
  cudaCheckError();
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);
  hipProfilerStop();

  // Release memory  
  for(int i = 0; i < NITERATIONS; i++){
    cipher_free(cipher, ct1[i]);
    cipher_free(cipher, ct3[i]);
  }
    
  keys_free(cipher, keys);
  poly_free(cipher, &sk->s);
  delete sk;
  delete keys;
  delete cipher;
  return latency / NITERATIONS;
}

__host__ double runMulPlain(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  std::default_random_engine generator;
  std::uniform_real_distribution<double> distribution;

  float latency = 0;

  // Setup
  CKKSContext *cipher = new CKKSContext();
  Sampler::init(cipher);
  SecretKey *sk = ckks_new_sk(cipher);
  Keys *keys = ckks_keygen(cipher, sk);

  complex<double> val1 = {distribution(generator), distribution(generator)};
  double val2 = distribution(generator);

  std::vector<cipher_t*> ct1;
  std::vector<cipher_t*> ct3;
  
  for(int i = 0; i < NITERATIONS; i++){
    ct1.push_back(ckks_encrypt(cipher, &val1));
    cipher_t *c = new cipher_t;
    cipher_init(cipher, c);
    ct3.push_back(c);
  }

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    ckks_mul(cipher, ct3[i], ct1[i], val2);

  // Benchmark
  hipDeviceSynchronize();
  cudaCheckError();
  hipProfilerStart();
  hipEventRecord(start, cipher->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
      ckks_mul(cipher, ct3[i], ct1[i], val2);  

  hipEventRecord(stop, cipher->get_stream());
  cudaCheckError();
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);
  hipProfilerStop();

  // Release memory  
  for(int i = 0; i < NITERATIONS; i++){
    cipher_free(cipher, ct1[i]);
    cipher_free(cipher, ct3[i]);
  }
    
  keys_free(cipher, keys);
  poly_free(cipher, &sk->s);
  delete sk;
  delete keys;
  delete cipher;
  return latency / NITERATIONS;
}

__host__ double runEnc(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  std::default_random_engine generator;
  std::uniform_real_distribution<double> distribution;

  float latency = 0;

  // Setup
  CKKSContext *cipher = new CKKSContext();
  Sampler::init(cipher);
  SecretKey *sk = ckks_new_sk(cipher);
  Keys *keys = ckks_keygen(cipher, sk);

  std::vector<poly_t*>   ms(NITERATIONS);
  std::vector<cipher_t> cts(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++){
    ms[i] = new poly_t;
    poly_init(cipher, ms[i]);
    cipher_init(cipher, &cts[i]);
  }

  // Warm-up
  ckks_encrypt_poly(cipher, &cts[0], ms[0]);

  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start, cipher->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    ckks_encrypt_poly(cipher, &cts[i], ms[i]);
  hipEventRecord(stop, cipher->get_stream());

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  // Release memory  
  for(int i = 0; i < NITERATIONS; i++){
    poly_free(cipher, ms[i]);
    cipher_free(cipher, &cts[i]);
  }
    
  keys_free(cipher, keys);
  poly_free(cipher, &sk->s);
  delete sk;
  delete keys;
  delete cipher;
  return latency / NITERATIONS;
}


__host__ double runDec(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  std::default_random_engine generator;
  std::uniform_real_distribution<double> distribution;

  float latency = 0;

  // Setup
  CKKSContext *cipher = new CKKSContext();
  Sampler::init(cipher);
  SecretKey *sk = ckks_new_sk(cipher);
  Keys *keys = ckks_keygen(cipher, sk);

  complex<double> m = {distribution(generator), distribution(generator)};
  std::vector<poly_t*> pts;
  std::vector<cipher_t*> cts;
  
  for(int i = 0; i < NITERATIONS; i++){
    cts.push_back(ckks_encrypt(cipher, &m));
    poly_t *pt = new poly_t;
    poly_init(cipher, pt);
    pts.push_back(pt);
  }
  // Warm-up
  ckks_decrypt_poly(cipher, pts[0], cts[0], sk);

  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start, cipher->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    ckks_decrypt_poly(cipher, pts[i], cts[i], sk);
  hipEventRecord(stop, cipher->get_stream());

  // 
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  // Release memory  
  for(std::vector<cipher_t*>::iterator it = cts.begin(); it != cts.end(); ++it)
    cipher_free(cipher, *it);
  for(std::vector<poly_t*>::iterator it = pts.begin(); it != pts.end(); ++it)
    poly_free(cipher, *it);

    
  keys_free(cipher, keys);
  poly_free(cipher, &sk->s);
  delete sk;
  delete keys;
  delete cipher;
  return latency / NITERATIONS;
}

__host__ double runModUp(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  Sampler::init(ctx);
  std::vector<poly_t> a(NITERATIONS);
  std::vector<poly_t> b(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++){
    poly_init(ctx, &a[i]);
    poly_init(ctx, &b[i], QBBase);
    Sampler::sample(ctx, &a[i], DISCRETE_GAUSSIAN);
  }

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    poly_modup(ctx, &b[i], &a[i], CUDAEngine::get_n_residues(QBase)-1);

  // Benchmark
  hipDeviceSynchronize();
  hipProfilerStart();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    poly_modup(ctx, &b[i], &a[i], CUDAEngine::get_n_residues(QBase)-1);
  hipEventRecord(stop, ctx->get_stream());
  hipProfilerStop();

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++){
    poly_free(ctx, &a[i]);
    poly_free(ctx, &b[i]);
  }
  delete ctx;
  return latency / NITERATIONS;
}

__host__ double runModDown(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  Sampler::init(ctx);
  std::vector<poly_t> a(NITERATIONS);
  std::vector<poly_t> b(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++){
    poly_init(ctx, &a[i], QBBase);
    poly_init(ctx, &b[i], QBase);
    Sampler::sample(ctx, &a[i], DISCRETE_GAUSSIAN);
  }

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    poly_moddown(ctx, &b[i], &a[i], CUDAEngine::get_n_residues(QBase)-1);

  // Benchmark
  hipDeviceSynchronize();
  hipProfilerStart();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    poly_moddown(ctx, &b[i], &a[i], CUDAEngine::get_n_residues(QBase)-1);
  hipEventRecord(stop, ctx->get_stream());
  hipProfilerStop();

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++){
    poly_free(ctx, &a[i]);
    poly_free(ctx, &b[i]);
  }
  delete ctx;
  return latency / NITERATIONS;
}

__host__ double runDiscreteGaussianSampler(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  Sampler::init(ctx);
  std::vector<poly_t> a(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++)
    poly_init(ctx, &a[i]);

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    Sampler::sample(ctx, &a[0], DISCRETE_GAUSSIAN);

  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    Sampler::sample(ctx, &a[i], DISCRETE_GAUSSIAN);
  hipEventRecord(stop, ctx->get_stream());

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++)
    poly_free(ctx, &a[i]);
  delete ctx;
  return latency / NITERATIONS;
 }

__host__ double runBinarySampler(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  Sampler::init(ctx);
  std::vector<poly_t> a(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++)
    poly_init(ctx, &a[i]);

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    Sampler::sample(ctx, &a[0], BINARY);

  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    Sampler::sample(ctx, &a[i], BINARY);
  hipEventRecord(stop, ctx->get_stream());

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++)
    poly_free(ctx, &a[i]);
  delete ctx;
  return latency / NITERATIONS;
 }


__host__ double runNarrowSampler(){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float latency = 0;

  // Setup
  Context *ctx = new Context();
  Sampler::init(ctx);
  std::vector<poly_t> a(NITERATIONS);
  for(int i = 0; i < NITERATIONS; i++)
    poly_init(ctx, &a[i]);

  // Warm-up
  for(int i = 0; i < NITERATIONS; i++)
    Sampler::sample(ctx, &a[0], NARROW);

  // Benchmark
  hipDeviceSynchronize();
  hipEventRecord(start, ctx->get_stream());
  for(int i = 0; i < NITERATIONS; i++)
    Sampler::sample(ctx, &a[i], NARROW);
  hipEventRecord(stop, ctx->get_stream());

  //
  hipEventSynchronize(stop);
  cudaCheckError();
  hipEventElapsedTime(&latency, start, stop);

  for(int i = 0; i < NITERATIONS; i++)
    poly_free(ctx, &a[i]);
  delete ctx;
  return latency / NITERATIONS;
 }

template <class T>
std::vector<T> intersection(std::vector<T> &v1,
                                      std::vector<T> &v2){
    std::vector<T> v3;

    std::sort(v1.begin(), v1.end());
    std::sort(v2.begin(), v2.end());

    std::set_intersection(v1.begin(),v1.end(),
                          v2.begin(),v2.end(),
                          back_inserter(v3));
    return v3;
}

 int main(int argc, char* argv[]){
  std::cout << "Benchmark: " << GET_AOADGT_VERSION() << std::endl;
  hipProfilerStop();

  /////////////////////////
  // Command line parser //
  ////////////////////////
  cxxopts::Options options("aoadgt_benchmark", "This program benchmarks the main procedures of SPOG-CKKS");
  options.add_options()
  ("p,procedure", "define which procedure should be measured", cxxopts::value<std::vector<std::string>>())
  ("d,degree", "for which ring degree it should measure the latencies", cxxopts::value<std::vector<int>>())
  ("r,residues", "how many residues should be instantiated", cxxopts::value<int>())
  ("h,help", "Print help and exit.")
  ;
  auto result = options.parse(argc, argv);

  // help
  if (result.count("help")) {
    cout << options.help({""}) << std::endl;
    exit(0);
  }

  srand(0);

  // Output precision
  cout << fixed;
  cout.precision(4);    
  Logger::getInstance()->set_mode(QUIET);  

  std::vector<std::string> type_data = {
    "Initialization",
    "CudaStreamInit",
    "PolyAdd",
    "PolyMul",
    "PolyMulAdd",
    "PolyDoubleAdd",
    "Encrypt",
    "Decrypt",
    "Add",
    "Mul",
    "AddPlain",
    "MulPlain",
    "Rescale",
    "Rotate",
    "Sumslots",
    "DGT", "IDGT",
    "ModUp", "ModDown"
  };
  std::vector<int> degrees_data = {
    2048,
    4096,
    8192,
    16384,
    32768,
    65536
  };
  std::vector<std::vector<double>> data(type_data.size(), std::vector<double>());

  // Select procedures
  if (result.count("procedure")){
    std::vector<std::string> v = result["procedure"].as<std::vector<std::string>>();
    type_data = intersection(type_data, v);
  }

  // Select ring degrees
  if (result.count("degree")){
    std::vector<int> v = result["degree"].as<std::vector<int>>();
    degrees_data = intersection(degrees_data, v);
  }

  std::map<int, int> parameters;
  if(result.count("gpu")){
    if(result["gpu"].as<std::string>().compare("k80") == 0){
      parameters[2048] = 2;
      parameters[4096] = 3;
      parameters[8192] = 6;
      parameters[16384] = 12;
      parameters[32768] = 10;
      parameters[65536] = 10;
    }else if(result["gpu"].as<std::string>().compare("v100") == 0){
      parameters[2048] = 2;
      parameters[4096] = 2;
      parameters[8192] = 2;
      parameters[16384] = 6;
      parameters[32768] = 10;
      parameters[65536] = 10;
    }
  }

  if(parameters.size() == 0){
    // Load k80 parameters
      int default_k;
      if(result.count("residues"))
        default_k = result["residues"].as<int>();
      else
        default_k = 2;
      parameters[2048] = default_k;
      parameters[4096] = default_k;
      parameters[8192] = default_k;
      parameters[16384] = default_k;
      parameters[32768] = default_k;
      parameters[65536] = default_k;
  }
  float latency = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  for(std::vector<int>::iterator d = degrees_data.begin(); d != degrees_data.end(); d++){
    // Init
    // 
    // Values:
    // 
    CUDAParams p;
    p.k = parameters[*d];
    p.kl = p.k+1;
    p.nphi = (*d);
    p.pt = 55;

    std::cout << p.k << ", " << p.kl << ", " << p.nphi << "< " << p.pt << std::endl;
    
    // Start the engine
    hipEventRecord(start);
    CUDAEngine::init(p);
    hipEventRecord(stop);
    cudaCheckError();
    hipEventSynchronize(stop);
    cudaCheckError();
    hipEventElapsedTime(&latency, start, stop);
    std::cout << "Initialization done in " << latency << std::endl;

    ZZ_p::init(CUDAEngine::RNSProduct);

    std::cout << *d << " (" << NTL::NumBits(CUDAEngine::RNSProduct) << " bits)" << std::endl;
    print_memory_usage();
    // Tests
    for(std::vector<std::string>::iterator it = type_data.begin(); it != type_data.end(); it++){
      double diff;
      if(*it == "Initialization")
        diff = runInit();
      else if (*it == "CudaStreamInit")
        diff = runCudaStreamInit();
      else if(*it == "Encrypt")
        diff = runEnc();
      else if (*it == "Decrypt")
        diff = runDec();
      else if (*it == "Add")
        diff = runAdd();
      else if (*it == "Mul")
        diff = runMul();
      else if (*it == "AddPlain")
        diff = runAddPlain();
      else if (*it == "MulPlain")
        diff = runMulPlain();
      else if (*it == "Rescale")
        diff = runRescale();
      else if (*it == "Rotate")
        diff = runRotate();
      else if (*it == "Sumslots")
        diff = runSumslots();
      else if (*it == "DGT")
        diff = runDGT();
      else if (*it == "IDGT")
        diff = runIDGT();
      else if (*it == "ModUp")
        diff = runModUp();
      else if (*it == "ModDown")
        diff = runModDown();
      else if (*it == "PolyAdd")
        diff = runPolyAdd();
      else if (*it == "PolyMul")
        diff = runPolyMul();
      else if (*it == "PolyMulAdd")
        diff = runPolyMulAdd();
      else if (*it == "PolyDoubleAdd")
        diff = runPolyAddAdd();
      else if (*it == "DiscreteGaussianSampler")
        diff = runDiscreteGaussianSampler();
      else if (*it == "BinarySampler")
        diff = runBinarySampler();
      else if (*it == "NarrowSampler")
        diff = runNarrowSampler();
      else 
        continue;

      data[distance(type_data.begin(), it)].push_back(diff);
    }

    // Release
    hipDeviceSynchronize();
    cudaCheckError();

    CUDAEngine::destroy();
    Sampler::destroy();
    cudaCheckError();
  }
  
      // 
  // Output
  // 
  const char separator    = ' ';
  const int nameWidth     = 10;
  const int numWidth      = 30;

  // Print degrees
  std::cout << std::left << setw(numWidth) << "Placeholder" << setw(nameWidth) << setfill(separator);
  for(std::vector<int>::iterator d = degrees_data.begin(); d != degrees_data.end(); d++)
    std::cout << (*d) << setw(nameWidth) << setfill(separator);
  std::cout << std::endl;

  // Print data
  for(std::vector<std::string>::iterator it = type_data.begin(); it != type_data.end(); it++){
    // Type name
    std::cout << std::left << setw(numWidth) << (*it) << setw(nameWidth) << setfill(separator);

    // Values
    std::vector<double> v = data[distance(type_data.begin(), it)];
    for(std::vector<double>::iterator t = v.begin(); t != v.end(); t++)
      std::cout << (*t) << setw(nameWidth) << setfill(separator);
    std::cout << std::endl;
  }

  hipDeviceReset();
  cudaCheckError();
}
